#include "hip/hip_runtime.h"
%%cu

#include <bits/stdc++.h>

using namespace std;

__global__ void add(int* A,int *B,int *C, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid < n)
    {
        C[tid] = A[tid] + B[tid];
    }
}

void initialize(int *a,int n)
{
    for (int i = 0; i < n;i++)
    {
        a[i] = rand() % 100;
    }
}

void print(int *a,int n)
{
    for (int i = 0; i < n;i++)
    {
        cout << a[i]<<" ";
    }
    cout << endl;
}

int main()
{
    int n = 5;

    int *A, *B, *C;

    int size = n;
    size_t memBytes = size * sizeof(int);

    A = new int[size];
    B = new int[size];
    C = new int[size];

    initialize(A, n);
    initialize(B, n);

    cout << "Vector A:- " << endl;
    print(A, n);
    cout << "Vector B:- " << endl;
    print(B, n);

    int *X, *Y, *Z;
    hipMalloc(&X, memBytes);
    hipMalloc(&Y, memBytes);
    hipMalloc(&Z, memBytes);

    hipMemcpy(X, A, memBytes, hipMemcpyHostToDevice);
    hipMemcpy(Y, B, memBytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    add<<<blocksPerGrid , threadsPerBlock>>>(X,Y,Z,n);

    hipMemcpy(C, Z, memBytes, hipMemcpyDeviceToHost);

    cout<<"Vector C :- "<<endl;
    print(C,n);

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);
}

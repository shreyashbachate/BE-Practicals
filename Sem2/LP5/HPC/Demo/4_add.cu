#include "hip/hip_runtime.h"
%%cu 
#include<bits/stdc++.h>
#include<hip/hip_runtime.h>

using namespace std;

__global__ void add(int *a,int *b,int *c,int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid<n)
    {
        c[tid] = a[tid] + b[tid];
    }
}

void initialize(int *a,int n)
{

    for (int i = 0; i < n;i++)
    {
        a[i] = rand() % 100;
    }
}

void print(int *a,int n)
{

    for (int i = 0; i < n;i++)
    {
        cout<<a[i]<<" ";
    }
}

int main()
{
    int n = 5;

    int size = n;
    size_t mem = size * sizeof(int);

    int *A, *B, *C;
    int *X, *Y, *Z;

    A = new int[size];
    B = new int[size];
    C = new int[size];

    initialize(A, n);
    initialize(B, n);

    cout<<"Vector A:- \n";
    print(A, n);

    cout<<"Vector B:- \n";
    print(B, n);

    hipMalloc(&X, mem);
    hipMalloc(&Y, mem);
    hipMalloc(&Z, mem);

    hipMemcpy(X, A, mem, hipMemcpyHostToDevice);
    hipMemcpy(Y, B, mem, hipMemcpyHostToDevice);

    int threads = 256;
    int blocksPerThread = (n + threads - 1) / threads;

    add<<<blocksPerThread,threads>>> (X,Y,Z,n);
    
    hipMemcpy(C, Z, mem, hipMemcpyDeviceToHost);

    cout<<"Vector C:- ";
    print(C, n);

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);

}
#include "hip/hip_runtime.h"
%%cu
#include <iostream>
using namespace std;

__global__ void add(int* A, int* B, int* C, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        C[tid] = A[tid] + B[tid];
    }
}


void initialize(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        vector[i] = rand() % 10;
    }
}

void print(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        cout << vector[i] << " ";
    }
    cout << endl;
}

int main() {
    int N = 4;
    int* A, * B, * C;

    int vectorSize = N;
    size_t vectorBytes = vectorSize * sizeof(int);

    A = new int[vectorSize];
    B = new int[vectorSize];
    C = new int[vectorSize];

    initialize(A, vectorSize);
    initialize(B, vectorSize);

    cout << "Vector A: ";
    print(A, N);
    cout << "Vector B: ";
    print(B, N);

    int* X, * Y, * Z;
    hipMalloc(&X, vectorBytes);
    hipMalloc(&Y, vectorBytes);
    hipMalloc(&Z, vectorBytes);

    hipMemcpy(X, A, vectorBytes, hipMemcpyHostToDevice);
    hipMemcpy(Y, B, vectorBytes, hipMemcpyHostToDevice);

    // hipEvent_t start,stop;
    // float elapsedTime;

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // hipEventRecord(start,0);

    add<<<blocksPerGrid, threadsPerBlock>>>(X, Y, Z, N);

    // hipEventRecord(stop,0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&elapsedTime,start,stop);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);

    hipMemcpy(C, Z, vectorBytes, hipMemcpyDeviceToHost);

    cout << "Addition: ";
    print(C, N);

    // cout<<"Elapsed Time = "<<elapsedTime<<" milliseconds" << endl;

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    return 0;
}

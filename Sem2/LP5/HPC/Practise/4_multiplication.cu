#include "hip/hip_runtime.h"
%%cu
#include<bits/stdc++.h>
#include<hip/hip_runtime.h>

#define BLOCK_SIZE 16

using namespace std;

__global__ void multiply(int *a,int *b,int *c,int c_rows,int common,int c_cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;

    if(col < c_cols && row < c_rows)
    {
        for (int j = 0; j < common;j++)
        {
            sum += a[row * common + j] * b[j * c_cols + col];
        }
        c[c_cols * row + col] = sum;
    }
}

void initialize(int *a,int rows,int cols)
{
    for (int i = 0; i < rows;i++)
    {
        for (int j = 0; j < cols;j++)
        {
            a[i * cols + j] = rand() % 10;
        }
    }
}

void print(int *a,int rows,int cols)
{
    for (int i = 0; i < rows;i++)
    {
        for (int j = 0; j < cols;j++)
        {
            cout << a[i * cols + j] << " ";
        }
        cout << endl;
    }
}



int main()
{
    int A_rows = 3, A_cols = 2, B_rows = 2, B_cols = 4, C_rows = A_rows, C_cols = B_cols;

    int *A, *B, *C;
    int *m1, *m2, *m3;

    int A_size = A_rows * A_cols;
    int B_size = B_rows * B_cols;
    int C_size = C_rows * C_cols;

    A = new int[A_size];
    B = new int[B_size];
    C = new int[C_size];

    hipMalloc(&m1, A_size * sizeof(int));
    hipMalloc(&m2, B_size * sizeof(int));
    hipMalloc(&m3, C_size * sizeof(int));


    initialize(A, A_rows, A_cols);
    cout << "Matrix A:- " << endl;
    print(A, A_rows, A_cols);

    initialize(B, B_rows, B_cols);
    cout << "Matrix B:- " << endl;
    print(B, B_rows, B_cols);

    hipMemcpy(m1, A, A_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(m2, B, B_size * sizeof(int), hipMemcpyHostToDevice);

    dim3 dimGrid(A_rows + BLOCK_SIZE - 1 / BLOCK_SIZE, B_cols + BLOCK_SIZE - 1 / BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    multiply<<<dimGrid, dimBlock>>>(m1, m2, m3, C_rows, A_cols, C_cols);

    hipMemcpy(C,m3,C_size*sizeof(int),hipMemcpyDeviceToHost);

    cout << "Matrix C:- " << endl;
    print(C, C_rows, C_cols);

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(m1);
    hipFree(m2);
    hipFree(m3);
}

